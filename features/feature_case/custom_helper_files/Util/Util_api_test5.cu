
#include <hip/hip_runtime.h>
// ====------ Util_api_test5.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Util/api_test5_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Util/api_test5_out/MainSourceFiles.yaml | wc -l > %T/Util/api_test5_out/count.txt
// RUN: FileCheck --input-file %T/Util/api_test5_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Util/api_test5_out

// CHECK: 2
// TEST_FEATURE: Util_vectorized_min

__device__ void foo() {
  unsigned u, u2;
  u = __vminu2(u, u2);
  u = __vminu4(u, u2);
}

int main() {
  return 0;
}
